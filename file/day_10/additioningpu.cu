#include "hip/hip_runtime.h"

#include <stdio.h>

// CUDA kernel to add two integers
__global__ void addIntegers(int* a, int* b, int* result) {
    *result = *a + *b;
}

int main(){
    //  Host variables
    int host_a = 5;
    int host_b = 7;
    int host_result = 0;

    // Device variables
    int* device_a, * device_b, * device_result;

    // Allocate memory on the device
    hipMalloc((void**)&device_a, sizeof(int));
    hipMalloc((void**)&device_b, sizeof(int));
    hipMalloc((void**)&device_result, sizeof(int));

    //Copy data from host to device
    hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with one block and one thread
    addIntegers <<<1,1>>> (device_a, device_b, device_result);

    // Copy the result from device to host
    hipMemcpy(&host_result, device_result, sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    printf("Sum of %d and %d is %d\n", host_a, host_b, host_result);

    // Free allocated memory
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);
    return 0;
}