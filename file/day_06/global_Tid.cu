#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 4
int rows = 10;
int cols = 10;
dim3 dimBlock(N,N);
dim3 dimGrid(rows + N - 1 / N, cols + N - 1 / N);

__global__ void GlobalId()
{
    int tidX = threadIdx.x + blockIdx.x * blockDim.x;
    int tidY = threadIdx.y + blockIdx.y * blockDim.y;
    int tBlocks = N*N;
    int Gid = tidX * tBlocks + tidY;
    printf("Global TID: %d\n", Gid);
}

int main()
{
    GlobalId<<<dimGrid,dimBlock>>>();
    hipDeviceSynchronize();
    return 0;
}