#include "hip/hip_runtime.h"
// Assignment 2 : From the original copy of the code, create a new code file. Try to make the square function also global,
// instead of it being a device function. Try to execute and note your observations.


#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ int square(int a) {
    // printf("Thread (%d, %d) - squaring value \n", blockIdx.x, threadIdx.x);
    a = a * a;
}

__global__ void doubleValues(int* data, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (i < size) {
        int value = data[i];
        square<<<1,1>>>(value);
        // Call square function and print thread ID
        int squared_value = square(value);

        // printf("Thread (%d, %d) - doubling squared value\n", blockIdx.x, threadIdx.x);
        data[i] = squared_value * 2;

         hipDeviceSynchronize();
    }return
}

int main() {
    // Allocate memory on host and device
    int size = 10000000;
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    // Initialize data on host
    for (int i = 0; i < size; ++i) {
        data_host[i] = i;
    }

    // Copy data to device
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);



   // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    // Launch kernel
    int threadsPerBlock = 256;
    doubleValues<<<(size + threadsPerBlock -1) / threadsPerBlock, threadsPerBlock>>> (data_device, size);

    // Wait for kernel to finish
    hipDeviceSynchronize();



// Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);



    // Copy data back from device
    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    // for ( int i= 0; i < size; ++i) {
    //     printf("data[%d] = %d\n", i, data_host[i]);
    // }



    // Print time taken by GPU
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);

    // Free memory
    hipFree(data_device);
    delete[] data_host;

    return 0;
    }

    // Time taken by GPU WITHOUT PRINTING : 0.149376 milliseconds