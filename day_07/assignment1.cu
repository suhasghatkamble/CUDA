//Assignment ( 25 may ) : Dynamic Parallelism - Maintain a copy of the original code. Modify the code to have the square functionality inside the doubleValue function itself.
// Have 1000000 elements, instead of 10 in both the versions. Compare the performance of the two versions by adding timing code.


#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void doubleValues(int* data, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {        
        data[i] = (data[i] * data[i]) * 2;
    }
}

int main() {
    // Allocate memory on host and device
    int size = 10000000;
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    // Initialize data on host
    for (int i = 0; i < size; ++i) {
        data_host[i] = i;
    }

    // Copy data to device
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);



   // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    int threadsPerBlock = 256;
    doubleValues<<<(size + threadsPerBlock -1) / threadsPerBlock, threadsPerBlock>>> (data_device, size);

    // Wait for kernel to finish
    hipDeviceSynchronize();


// Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    



    // Copy data back from device
    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    // // Print results
    // for ( int i= 0; i < size; ++i) {
    //     printf("data[%d] = %d\n", i, data_host[i]);
    // }

    // Print time taken by GPU
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);


    // Free memory
    hipFree(data_device);
    delete[] data_host;

    return 0;
    }


// Time taken by GPU WITH PRINTING VALUES : 8.354336 milliseconds


// Time taken by GPU : 0.152288 milliseconds
//Time taken by GPU : 0.153952 milliseconds
//Time taken by GPU : 0.153376 milliseconds
// Time taken by GPU : 0.166208 milliseconds
