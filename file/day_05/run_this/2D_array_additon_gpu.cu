#include "hip/hip_runtime.h"
// Assignment : Modify the array addition example to make it two dimensional.


#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_arrays(int *c, const int *a, const int *b, int size) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // finding thread number
    int j = blockIdx.y * blockDim.y + threadIdx.y;


    if(i < size && j< size){
    c[i][j] = a[i][j] + b[i][j];                       //check it checks whether size is not greater than array size
    }
}

int main() {
    const int size = 5;

    int a[size] = {1 , 2 , 3 , 4 , 5};
    int b[size] = {1 , 2 , 3 , 4 , 5};
    int *d_c;

    // Allocate memory on the device for array c 

    hipMalloc((void**)&  d_c, size * sizeof(int));

    // Copy array a and b to the device

    int *d_a, *d_b;

    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);



    dim3 threadsPerBlock(16,16);
    dim3 numBlocks(((N + threadsPerBlock.x - 1) / threadsPerBlock.x) , ((N + threadsPerBlock.y -1) / threadsPerBlock.y));
    
    // Launch kernel
    add_arrays<<<(125,125), (16,16)>>>(d_c, d_a, d_b, size);
    hipDeviceSynchronize();

    // Copy the result back from the device
    int *c = (int*)malloc(5 * sizeof(int));
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("%d ", c[i]);
    }
    
    printf("\n");
     
     // Free memory on the device
     hipFree(c);
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);

     return 0;
}





