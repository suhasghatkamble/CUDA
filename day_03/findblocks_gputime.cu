#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add_arrays(int *c, const int *a, const int *b, int size) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // finding thread number
    
    if(i < size){

	c[i] = a[i] + b[i];                       
	}
}

int main() {
    const int size = 90000;

    int *d_c;

    int a[size];
    int b[size];
	

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory on the device for array c 

    hipMalloc((void**)&  d_c, size * sizeof(int));

    // Copy array a and b to the device

    int *d_a, *d_b;

    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);


    // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Calculating threads and blocks per grid
    int threadsPerBlock=1024;
    int blocksPerGrid = (size + threadsPerBlock - 1)/ threadsPerBlock;

    // passing threadsperblocks and blockspergrid
    add_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);
    hipDeviceSynchronize();

    // Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result back from the device
    int *c = (int*)malloc(size * sizeof(int));
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("a = %d ,b = %d, c = %d \n",a[i],b[i], c[i]);
    }
    
    printf("\n");
    
    // Print time taken by GPU
    printf("Time taken by GPU : %f milliseconds\n", milliseconds);
     
     // Free memory on the device
     hipFree(c);
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);

     return 0;
}

//0.025792 time taken

// TO RUN CODE

// nvcc findblocks_gputime.cu
// ./a.out



// cpu                     gpu
// a[1000]  --copy--       a[]
// b[1000]  --copy--       b[]
// c[]                     c[a+b] ----- copy to cpu c[]