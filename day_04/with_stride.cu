#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectorAddition_kernel(int* d_a, int* d_b, int* d_c, int N) {
    int tid = threadIdx.x;
    int i;
    for (i = tid; i < N; i+= blockDim.x) {
        d_c[i] = d_a[i] + d_b[i];
        printf("\n Threadid : %d = %d\n", tid,d_c[i]);
    }
}

int main() {
    int N = 12;
    int* h_a, * h_b, * h_c; // Host variable
    int* d_a, * d_b, * d_c; // Device variable

    // Allocate memory for host variable
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Allocate memory for device variables
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Initialize host variables
    for (int i=0; i < N; i++){
        h_a[i] = 2;
        h_b[i] = 2;
        h_c[i] = 0;
    }

    // Copy host variables to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);


    // Launch kernel function
    int blockSize = 1;
    int numThreads = 4;
    vectorAddition_kernel <<<blockSize, numThreads>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display results
    printf("Result: ");
    for (int i = 0; i< N; i++) {
        printf("%d",h_c[i]);
    }
    printf("\n");

    // Free device and host Memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;

    // here 1 block and 8 threads




    

}
