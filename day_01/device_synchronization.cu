// importance of function call in cuda


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void) {
	printf("Hello from GPU \n");
}

void cpu_print(void) {
	printf("Hello from CPU  \n");
}

int main() {
	kernel <<<1, 1>>> ();
	kernel <<<1, 1>>> ();
	kernel <<<1, 1>>> ();

	hipDeviceSynchronize();
	//wait till gpu work done , for synchronization
	//gate betw cpu and gpu, cpu cant go next before execution of gpu work
	//telling the cpu to wait till all the gpu work complete then only cpu work 

	cpu_print();
	cpu_print();
	cpu_print();

	return 0;
}


