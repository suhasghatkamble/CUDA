#include<stdio.h>
#include<hip/hip_runtime.h>


__device__ int square(int a) {
    printf("Thread (%d, %d) - squaring value \n", blockIdx.x, threadIdx.x);
    return a * a;
}

__global__ void doubleValues(int* data, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int value = data[i];
        // Call square function and print thread ID
        int squared_value = square(value);

        printf("Thread (%d, %d) - doubling squared value\n", blockIdx.x, threadIdx.x);
        data[i] = squared_value * 2;
    }
}

int main() {
    // Allocate memory on host and device
    int size = 10;
    int* data_host = new int[size];
    int* data_device;
    hipMalloc(&data_device, size * sizeof(int));

    // Initialize data on host
    for (int i = 0; i < size; ++i) {
        data_host[i] = i;
    }

    // Copy data to device
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    doubleValues<<<(size + threadsPerBlock -1) / threadsPerBlock, threadsPerBlock>>> (data_device, size);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy data back from device
    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    for ( int i= 0; i < size; ++i) {
        printf("data[%d] = %d\n", i, data_host[i]);
    }

    // Free memory
    hipFree(data_device);
    delete[] data_host;

    return 0;
    }