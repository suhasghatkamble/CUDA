// Finding maximum number of threads and blocks on our GPU ...


#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int maxThreadsPerBlock, maxBlocks;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocks, hipDeviceAttributeMaxGridDimX, 0);

    std::cout << "Maximum Threads Per Block : " << maxThreadsPerBlock << std::endl;
    std::cout << "Maximum Blocks: " << maxBlocks << std::endl;

    return 0;
}


// To run code 

// nvcc find_max_threads_blocks.cu
// ./a.out