#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void add_arrays(int *c, const int *a, const int *b, int size) {

    int threadID = blockIdx.x * blockDim.x + threadIdx.x;  // finding thread number
    
    if(i<=100001)
    {
    int flag = 1
	for (size_t i = 3; i < 100001; i++)
    {
        if (threadID%1==0)
        {
            flag=0;
            break;
        }
        
    }                     
	}

    if(flag==1){
        printf("Prime number");
    }
    else{
        printf("Not prime Number")
    }
}

int main() {
    const int size = 100001;

    int *d_c;

    int a[size];
    int b[size];
	

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory on the device for array c 

    hipMalloc((void**)&  d_c, size * sizeof(int));

    // Copy array a and b to the device

    int *d_a, *d_b;

    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);


    // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Calculating threads and blocks per grid
    int threadsPerBlock=1024;
    int blocksPerGrid = (size + threadsPerBlock - 1)/ threadsPerBlock;

    // passing threadsperblocks and blockspergrid
    add_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);
    hipDeviceSynchronize();

    // Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result back from the device
    int *c = (int*)malloc(size * sizeof(int));
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("a = %d ,b = %d, c = %d \n",a[i],b[i], c[i]);
    }
    
    printf("\n");
    
    // Print time taken by GPU
    printf("Time taken by GPU : %f milliseconds\n", milliseconds);
     
     // Free memory on the device
     hipFree(c);
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);

     return 0;
}
