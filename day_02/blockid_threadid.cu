#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void printThreadInfo() {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    printf("ThreadIdx: %d, BlockId: %d, BlockDim: %d, Effective Thread ID: %d\n", threadIdx.x, blockIdx.x, blockDim.x, threadID);

}

int main() {
    int numBlocks = 3;
    int threadsPerBlock = 4;

    printThreadInfo<<<numBlocks, threadsPerBlock>>>();
    //or printThreadInfo<<<3, 4>>>();
    hipDeviceSynchronize();

    return 0;
}