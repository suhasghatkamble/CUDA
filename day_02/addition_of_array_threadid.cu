#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_arrays(int *c, const int *a, const int *b, int size) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // finding thread number
    
    if(i < size){
     c[i] = a[i] + b[i];                       //check it checks whether size is not greater than array size
    }
}

int main() {
    const int size = 5;

    int a[size] = {1 , 2 , 3 , 4 , 5};
    int b[size] = {1 , 2 , 3 , 4 , 5};
    int *d_c;

    // Allocate memory on the device for array c 

    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy array a and b to the device

    int *d_a, *d_b;

    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    add_arrays<<<2, 4>>>(d_c, d_a, d_b, size);
    hipDeviceSynchronize();

    // Copy the result back from the device
    int *c = (int*)malloc(5 * sizeof(int));
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("%d ", c[i]);
    }
    
    printf("\n");
     
     // Free memory on the device
     hipFree(c);
     hipFree(d_a);
     hipFree(d_b);
     hipFree(d_c);

     return 0;
}