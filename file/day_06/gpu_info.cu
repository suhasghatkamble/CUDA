#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 1;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        printf("Device %d: %s\n", device, deviceProp.name);
        printf(" Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf(" Total Global Memory: %lu bytes \n", (unsigned long) deviceProp.sharedMemPerBlock);
        printf(" Warp Size: %d\n", deviceProp.warpSize);
        printf(" Max Threads Per Block: %d\n", deviceProp.maxThreadsPerBlock);
        printf(" Max Threads Dimension: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf(" Max Grid Size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf(" Clock Rate: %d kHz\n", deviceProp.clockRate);
        printf(" Memory Clock Rate: %d kHz\n", deviceProp.memoryClockRate);
        printf(" Memory Bus Width: %d bits\n", deviceProp.memoryBusWidth);
        printf(" L2 Cache Size: %d bytes\n", deviceProp.l2CacheSize);
        printf(" Constant Memory Size: %lu bytes\n", (unsigned long) deviceProp.totalConstMem);
        printf(" Texture Alignment: %lu bytes\n", (unsigned long) deviceProp.textureAlignment);
        printf("\n");
            
        }
        return 0;
}