#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void) {
        printf("Hello from GPU \n");
}

void cpu_print(void) {
        printf("Hello from CPU  \n");
}

int main() {
        kernel <<<1, 10>>> (); // kernel <<< total_block, total_thread >>> ()

        hipDeviceSynchronize();
        
        cpu_print();
   
        return 0;
}

