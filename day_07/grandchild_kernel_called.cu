#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void grandchildKernel() {
    printf("From C-DAC ");
}

__global__ void childKernel() {

    grandchildKernel<<<1,1>>>();

    hipDeviceSynchronize();  // // Wait for the grandchild to complete 

    printf("Hello ");
}

__global__ void parentKernel() {
    childKernel<<<1,1>>>();

    hipDeviceSynchronize();  // Wait for the child to complete , first print child's

    printf("World!\n");
}

int main() {
    parentKernel<<<1,1>>>();

    hipDeviceSynchronize();  // Wait for the parent to complete

    return 0;
}

//nvcc -rdc=true grandchild_kernel_called.cu