#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void childKernel() {
    printf("Hello ");
}

__global__ void parentKernel() {
    childKernel<<<1,1>>>();

    hipDeviceSynchronize();  // Wait for the child to complete , first print child's

    printf("World!\n");
}

int main() {
    parentKernel<<<1,1>>>();

    hipDeviceSynchronize();  // Wait for the parent to complete

    return 0;
}

//nvcc -rdc=true global_called_global.cu