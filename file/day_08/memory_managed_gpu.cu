#include <stdio.h>
#include <hip/hip_runtime.h>
 



__global__ void printValue(int* data) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Access managed memory directly from the GPU
    printf("GPU Thread %d: value = %d\n", tid, data[tid]);
}

int main() {
    const int N = 10;

    // Allocated managed memroy 
    int* data;
    hipMallocManaged(&data, N * sizeof(int));

    // Initialize data on the CPU
    for (int i = 0; i < N; ++i) {
        data[i] = i * 2;
    }

    // Launch GPU kernel to print values
    printValue <<<1, N>>>(data);
    hipDeviceSynchronize();   // Ensure GPU kernel completes

    // Access managed memroy directly from the CPU
    for (int i = 0; i < N; ++i) {
        printf("CPU: Value = %d\n", data[i]);
    }

    // Free managed memory
    hipFree(data);

    return 0;
}