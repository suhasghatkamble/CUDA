#include<stdio.h>
#include<hip/hip_runtime.h>

//CUDA kernel function for printing prime number
__global__ void printPrimeNumbers(){
    int threadID=blockIdx.x * blockDim.x + threadIdx.x;

if(threadID>2 && threadID<=10000001)
{ 
  int flag=1;//it is prime number
    
    for (size_t i = 2; i < threadID; i++)
    {
        if(threadID%i==0){
            flag=0;//it it not a prime number
            break;
        }
    }

    if(flag)
    {
        // printf("%d ",threadID); //it will print threads which are prime
    }           
}

}





int main()
{
    // create host var
    int host_fromNum = 2;
    int host_toNum =10000001;


    //Declare for Device variables/pointers
    int *device_fromNum;
    int *device_toNum;

    //Allocate memory on the device/host
    hipMalloc((void**)&device_fromNum, sizeof(int));
    hipMalloc((void**)&device_toNum, sizeof(int));

    //copy data from host to device
    hipMemcpy(device_fromNum,&host_fromNum,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_toNum,&host_toNum,sizeof(int),hipMemcpyHostToDevice);

    int threadPerBlock=512;
    int blockPerGrid=(host_toNum+threadPerBlock-1)/threadPerBlock;

   // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    printPrimeNumbers <<<blockPerGrid,threadPerBlock>>>();
    hipDeviceSynchronize();


// Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print time taken by GPU
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);

    //free alloacted memory
    hipFree(device_fromNum);
    hipFree(device_toNum);


}