#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define NUM_CLASSES 3

// Function to be executed on the GPU (kernel)
__global__ void calculateComfort(int *d_classArray, float *d_seatComfortArray, float *d_totalComfort, int *d_classCounts, int numRecords) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < numRecords) {
    int classIndex = d_classArray[idx];
    float seatComfort = d_seatComfortArray[idx];

    atomicAdd(&d_totalComfort[classIndex], seatComfort);
    atomicAdd(&d_classCounts[classIndex], 1);
  }
}

int main() {
  // ... (same as original CPU code for file handling and data allocation)

  // Allocate device memory (GPU)
  int *d_classArray;
  float *d_seatComfortArray;
  float *d_totalComfort;
  int *d_classCounts;

  hipMalloc((void**)&d_classArray, numRecords * sizeof(int));
  hipMalloc((void**)&d_seatComfortArray, numRecords * sizeof(float));
  hipMalloc((void**)&d_totalComfort, NUM_CLASSES * sizeof(float));
  hipMalloc((void**)&d_classCounts, NUM_CLASSES * sizeof(int));

  // Copy data from host to device
  hipMemcpy(d_classArray, classArray, numRecords * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_seatComfortArray, seatComfortArray, numRecords * sizeof(float), hipMemcpyHostToDevice);

  // Initialize total comfort and class counts on device (zero them out)
  hipMemset(d_totalComfort, 0, NUM_CLASSES * sizeof(float));
  hipMemset(d_classCounts, 0, NUM_CLASSES * sizeof(int));

  auto start_time = std::chrono::high_resolution_clock::now();

  // Launch the kernel on the GPU
  int threadsPerBlock = 256;  // Adjust as needed for your GPU
  int blocks = (numRecords + threadsPerBlock - 1) / threadsPerBlock;
  calculateComfort<<<blocks, threadsPerBlock>>>(d_classArray, d_seatComfortArray, d_totalComfort, d_classCounts, numRecords);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  auto end_time = std::chrono::high_resolution_clock::now();

  // ... (same as original CPU code for calculating time taken and reporting results)

  // Free device memory
  hipFree(d_classArray);
  hipFree(d_seatComfortArray);
  hipFree(d_totalComfort);
  hipFree(d_classCounts);

  // ... (same as original CPU code for freeing host memory)

  return 0;
}